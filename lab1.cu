
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdint>
#include <math.h>
#include <chrono>
#include <climits>


// CPU section

bool CPUprime(uint64_t n){
    auto start = std::chrono::high_resolution_clock::now();

    // check 2 and 3
    if (n <= 3){
        return (n > 1);
    }
    // check if number isn't even
    if (n % 2 == 0){
        return false;
    }

    // loop through every six number less than sqrt(num) starting at 5
    for(int i = 5; i < std::ceil(std::sqrt(n)); i+=6) {
        if(n % i == 0 || n % (i +2) == 0)
        {
            auto stop = std::chrono::high_resolution_clock::now();

            auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
            std::cout << "Test duration: " << duration.count() << " microseconds ";
            return false; 
        }
    }

    auto stop = std::chrono::high_resolution_clock::now();

    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    std::cout << "Test duration: " << duration.count() << " microseconds ";

    return true;
}


void testCPU(bool (*func)(uint64_t)){
    // test numbers given in the task

    uint64_t num[6] = {524287, 2147483647, 2305843009213693951, 274876858369, 4611686014132420609, 1125897758834689 };
    bool res[6];

    std::cout << "CPU primality test:\n";
    auto start = std::chrono::high_resolution_clock::now();    

    for(int i = 0; i < 6; i++){
        res[i] = func(num[i]);
        std::cout << "Is number " << num[i] << " prime?: ";
        if(res[i] == 1){
            std::cout << " Yes it is" << std::endl;
        }
        else{
            std::cout << " No it isn't" << std::endl;

        }
    }

    auto stop = std::chrono::high_resolution_clock::now();

    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    std::cout << "CPU test duration: " << duration.count() << " microseconds" << std::endl;

}



// GPU section


__global__
void dev_GPUgridstride(uint64_t* num, uint32_t* res, uint32_t* maxind){
    // use grid-stride loop to reuse threads

    for(uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
        5 + i * 6 <= *maxind - 2;
        i += blockDim.x * gridDim.x )
        {
            uint32_t realindex = 5 + i * 6;

            if(*num % realindex == 0 || *num % (realindex+2) == 0)
            {
                *res = 0;
            }
        }
}

__global__ void dev_GPUmonolithic(uint64_t* num, uint32_t* res, uint32_t* maxind){

    uint32_t index = (threadIdx.x + blockIdx.x * blockDim.x);
    uint32_t realindex = 5 + index * 6;

    if(realindex <= *maxind-2){
        if(*num % realindex == 0 || *num % (realindex+2) == 0)
        {
            *res = 0;
        }
    }
}

bool GPUgridstride(uint64_t num, bool gridstride){

    auto start = std::chrono::high_resolution_clock::now();

    uint32_t sqrtnum = (uint32_t)std::floor(std::sqrt(num));
    uint32_t *res = new uint32_t;
    *res = 1;

    uint64_t* dnum;
    uint32_t* dres;
    uint32_t* maxind;


    // check 2 and 3
    if (num <= 3){
        return (num > 1);
    }
    // check if number isn't even
    if (num % 2 == 0){
        return false;
    }

    // allocate CUDA memory
    hipMalloc(&maxind, 4);
    hipMalloc(&dnum, 8);
    hipMalloc(&dres, 4);
    hipMemcpy(maxind, &sqrtnum, 4, hipMemcpyHostToDevice);
    hipMemcpy(dnum, &num, 8, hipMemcpyHostToDevice);
    hipMemcpy(dres, res, 4, hipMemcpyHostToDevice);

    if(gridstride == 1){

        // number of threads and blocks
        uint32_t threads = 1024;
        int numSMs;
        hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, 0);

        // kernel call
        dev_GPUgridstride<<<4*numSMs, threads>>>(dnum, dres, maxind);
        hipDeviceSynchronize();
    }
    else{
        uint32_t bitnum = ((sqrtnum - 5) / 6 + 1);

        dim3 blocksize= {32};
        dim3 gridsize = {bitnum/32 + (bitnum%32 !=0)};
        
        dev_GPUmonolithic<<<gridsize, blocksize>>>(dnum, dres, maxind);
        hipDeviceSynchronize();
    }

    // copy the solution and check it
    hipMemcpy(res, dres, 4, hipMemcpyDeviceToHost);

    bool prime = true;
    if(*res == 0){
        prime = false;
    }

    // deallocate the memory
    hipFree(dnum);
    hipFree(dres);
    hipFree(maxind);
    delete res;

    auto stop = std::chrono::high_resolution_clock::now();

    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    std::cout << "Test duration: " << duration.count() << " microseconds ";

    return prime;

}

// adding naive gpu



// bool GPUprime(uint64_t num){

//     uint32_t sqrtnum = (uint32_t)std::floor(std::sqrt(num));
//     uint32_t bitnum = ((sqrtnum - 5) / 6 + 1);
//     uint32_t reslen = (bitnum / 64 + (bitnum % 64 != 0)) * 2;
//     uint32_t *res = new uint32_t;

//     *res = 1;

//     uint64_t* dnum;
//     uint32_t* dres;
//     uint32_t* maxind;

//     auto start = std::chrono::high_resolution_clock::now();
//     cudaMalloc(&maxind, 4);
//     cudaMalloc(&dnum, 8);
//     cudaMalloc(&dres, reslen*4);
//     cudaMemcpy(maxind, &sqrtnum, 4, cudaMemcpyHostToDevice);
//     cudaMemcpy(dnum, &num, 8, cudaMemcpyHostToDevice);
//     cudaMemcpy(dres, res, reslen*4, cudaMemcpyHostToDevice);
//     dim3 blocksize= {32};
//     dim3 gridsize = {bitnum/32 + (bitnum%32 !=0)};
    
//     dev_GPUmonolithic<<<gridsize, blocksize>>>(dnum, dres, maxind);
//     cudaDeviceSynchronize();

//     cudaMemcpy(res, dres, reslen*4, cudaMemcpyDeviceToHost);
//     auto stop = std::chrono::high_resolution_clock::now();
    
//     bool prime = true;
//     for(int j = 0; j < reslen ;j++){
        
//         if(res[j] != UINT_MAX){
//             prime = false;
//         }
//     }

//     cudaFree(dnum);
//     cudaFree(dres);
//     cudaFree(maxind);
//     delete[] res;


//     auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(stop - start);
//     std::cout << "current number: " << duration.count() << " milliseconds" << std::endl;

//     return prime;

// }

// ///////////


void testGPU(bool (*func)(uint64_t, bool), bool gridstride = 1){
    // test numbers given in the task

    const int testlen = 6;
    uint64_t num[testlen] = {524287, 2147483647, 2305843009213693951, 274876858369, 4611686014132420609, 1125897758834689 };
    bool res[testlen];
    hipDeviceSynchronize();


    std::cout << "GPU primality test: ";
    if(gridstride == 1){
        std::cout << "grid-stride version" << std::endl;
    }
    else{
        std::cout << "monolithic kernel version" << std::endl;
    }
    auto start = std::chrono::high_resolution_clock::now();    

    for(int i = 0; i < testlen; i++){
        res[i] = func(num[i], gridstride);
        std::cout << "Is number " << num[i] << " prime?: ";
        if(res[i] == 1){
            std::cout << " Yes it is" << std::endl;
        }
        else{
            std::cout << " No it isn't" << std::endl;

        }
    }
    auto stop = std::chrono::high_resolution_clock::now();

    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(stop - start);
    std::cout << "GPU test duration: " << duration.count() << " microseconds" << std::endl;

}


int main() {

    // CPU test
    testCPU(&CPUprime);

    // Gpu test
    testGPU(&GPUgridstride);


}